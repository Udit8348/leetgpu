#include <hip/hip_runtime.h>

__global__ void vector_add(const float* A, const float* B, float* C, int N) {
    // we only need to sum the N requested elements in parallel
    // threadsPerBlock is not constrained to problem size, so we need this conditional
    
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < N) {
        C[thread_id] = B[thread_id] + A[thread_id];
    }
}

// A, B, C are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const float* A, const float* B, float* C, int N) {
    int threadsPerBlock = 512;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    vector_add<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, N);
    hipDeviceSynchronize();
}