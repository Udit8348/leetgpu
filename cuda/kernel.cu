#include "hip/hip_runtime.h"
#include <iostream>

// __global__ indicates this function should be compiled for the device (not host)
// nvcc routes this function to to the cuda compiler and main to the host's compiler
__global__ void kernel( void ) {
}

int main( void ) {
    int blocksPerGrid
    kernel<<<1,1>>>();
    printf( "Hello, World!\n" );
    return 0;
}